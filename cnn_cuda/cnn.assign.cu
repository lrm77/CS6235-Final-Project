#include "hip/hip_runtime.h"
#include "driver.cu"

inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<std::endl;
        exit(-1);
    }
}


int main(int argc, char *argv[]) {
    // READ PROBLEM SIZES
    if (argc != 10) exit(1);
    int N = atoi(argv[1]);
    int C = atoi(argv[2]);
    int K = atoi(argv[3]);
    int H = atoi(argv[4]);
    int W = atoi(argv[5]);
    int R = atoi(argv[6]);
    int S = atoi(argv[7]);
    int u = atoi(argv[8]);
    int v = atoi(argv[9]);
    int P = (H-R)/u + 1;
    int Q = (W-S)/v + 1;

    float *output_seq = new float[N*K*P*Q];
    memset(output_seq,0, N * K * P * Q*sizeof(float));
    float *output_par = new float[N*K*P*Q];
    memset(output_par,0, N * K * P * Q*sizeof(float));
    float *input = new float[N*C*H*W];
    float *weight = new float[K*C*R*S];
    // ASSIGN INITIAL VALUES FOR INPUT AND WEIGHT

    for(unsigned int n=0; n<N; ++n){
        for(unsigned int c=0; c<C; ++c){
            for(unsigned int h=0; h<H; ++h){
                for(unsigned int w=0; w<W; ++w){
                    input[n*C*H*W + c*H*W + h*W + w] =  ((float)(n+c+h+w));
                }
            }
        }
    }
    for (unsigned int k=0; k<K; k++) {
        for (unsigned int c=0; c<C; c++) {
            for (unsigned int r =0; r<R; r++) {
                for (unsigned int s =0; s<S; s++) {
                    //weight[k][c][r][s] = ((float) (k+c+r+s));
                    weight[k*C*R*S + c*R*S + r*S + s] = ((float) (k+c+r+s));
                }
            }
        }
    }
    // TIME SEQUENTIAL CALCULATION
    hipEvent_t seq_start,seq_stop;
    float seq_time;
    hipEventCreate(&seq_start);
    hipEventCreate(&seq_stop);
    hipEventRecord(seq_start);

    for(unsigned int n=0; n<N; n++) { // minibatch size
        for(unsigned int k=0; k<K; k ++) { // output feature map
            for(unsigned int c=0; c<C; c ++) { // input feature map
                for(unsigned int p=0; p<P; p ++) { // output height
                    unsigned int ij = p * u; // input height
                    for (unsigned int q = 0; q<Q; q ++) { // output width
                        unsigned int ii = q * v; // input width
                        for (unsigned int r = 0; r<R; r ++) { // filter height
                            for (unsigned int s = 0; s < S; s ++) {// filter width
                                //output_seq[n][k][p][q] += input [n][c][ij+r][ii+s] * weight[k][c][r][s];
                                output_seq[n*K*P*Q + k*P*Q + p*Q + q] += input[n*C*H*W + c*H*W + (ij+r)*W + ii+s] * weight[k*C*R*S+c*R*S+r*S+s];
                            }
                        }
                    }
                }
            }
        }
    }

    hipEventRecord(seq_stop);
    hipEventSynchronize(seq_stop);
    hipEventElapsedTime(&seq_time,seq_start, seq_stop);
    //@@ Copy input, weight and output data, input as example
    float * d_input, *d_weight, * d_output;
    chkerr(hipMalloc((void **) &d_input,  sizeof(float) * N * C * H * W));
    chkerr(hipMalloc((void **) &d_weight,  sizeof(float) * K * C * R * S));
    chkerr(hipMalloc((void **) &d_output,  sizeof(float) * N * K * P * Q));
    chkerr(hipMemcpy(d_input, input, sizeof(float) * N * C * H * W, hipMemcpyHostToDevice));
    chkerr(hipMemcpy(d_weight, weight, sizeof(float) * K * C * R * S, hipMemcpyHostToDevice));


    //@@ Launch the GPU Kernel here, you may want multiple implementations to compare
    dim3 dimBlock(64, 1, 1);
    dim3 dimGrid(ceil((P*Q)/(float)dimBlock.x), ceil(K/(float)dimBlock.y/8.0), ceil(N/(float)dimBlock.z));

    // INITIALIZE PARALLEL TIMER
    hipEvent_t par_start,par_stop;
    float par_time;
    hipEventCreate(&par_start);
    hipEventCreate(&par_stop); hipEventRecord(par_start);

    cnn<<<dimGrid, dimBlock>>>(N,C,K,H,W,R,S,u,v,P,Q,d_input,d_weight,d_output);

    hipEventRecord(par_stop);
    hipEventSynchronize(par_stop);
    hipEventElapsedTime(&par_time,par_start, par_stop);

    //@@ Copy the GPU memory back to the CPU here
    chkerr(hipMemcpy(output_par, d_output, sizeof(float)*N*K*P*Q, hipMemcpyDeviceToHost));

    //@@ Free the GPU memory here

    chkerr(hipFree(d_input));
    chkerr(hipFree(d_weight));
    chkerr(hipFree(d_output));

    // VERIFY CORRECTNESS BY COMPARING OUTPUTS
    for (unsigned int n=0; n<N; n++) { // minibatch size
        for (unsigned int k=0; k<K; k ++) { // output feature map
            for (unsigned int p=0; p<P; p ++) { // output height
                for (unsigned int q =0; q<Q; q ++) { // output width
                    if(abs(output_seq[n*K*P*Q+k*P*Q+p*Q+q]-output_par[n*K*P*Q+k*P*Q+p*Q+q])> .001) {
                        printf("Outputs do not match!!!\n");
                        // printf("n=%d, k=%d, p=%d, q=%d\n", n, k, p, q);
                        // printf("diff = %f\n", abs(output_seq[n*K*P*Q+k*P*Q+p*Q+q]-output_par[n*K*P*Q+k*P*Q+p*Q+q]));
                        // printf("seq = %f, par = %f\n", output_seq[n*K*P*Q+k*P*Q+p*Q+q], output_par[n*K*P*Q+k*P*Q+p*Q+q]);
                        exit(2);
                    }
                }
            }
        }
    }

    // PRINT OUT SPEEDUP
    printf ("Sequential time = %f, Parallel time = %f, Speedup = %f\n",seq_time, par_time, seq_time/par_time);
}
